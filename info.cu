
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

int main()
{
	int count;
	hipGetDeviceCount(&count);
	hipDeviceProp_t prop;

	for (int i = 0; i < count ; ++i )
	{
		hipGetDeviceProperties(&prop, i);
		cout << "Device : " << prop.name << endl;
		cout << "Compute Capability " << prop.major << " : "
				<< prop.minor << endl ;
		cout << "Grid Dimension " << prop.maxGridSize[0] << " x "
			 << prop.maxGridSize[1] << " x "<< prop.maxGridSize[2] << endl;

		cout << "Block Dimension " << prop.maxThreadsDim[0] << " x "
			 << prop.maxThreadsDim[1] << " x "<< prop.maxThreadsDim[2] << endl;
	}

	return 0;
}
